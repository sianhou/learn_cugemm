//
// Created by sianh on 2023/3/6.
//

#include "hip/hip_runtime.h"
#include "iostream"
#include "vector"
#include "iomanip"

__global__ void cu_sgemm(const float *a, const float *b, float *c, int M, int N, int K) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx < M && ty < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += a[tx * K + i] * b[i * K + ty];
        }
        c[tx * K + ty] = sum;
    }
}

template<int BLKX, int BLKY>
class TestGemm {
public:
    TestGemm(int M, int N, int K) : M(M), N(N), K(K) {
        block.x = BLKX;
        block.y = BLKY;
        grid.x = (M + BLKX - 1) / BLKX;
        grid.y = (N + BLKY - 1) / BLKY;
    }

    void WarmUp();
    float Run(int n_iter);
    void InitHost();
    void InitDevice();
    void Free();

    int M, N, K;
    dim3 grid, block;
    float *d_A, *d_B, *d_C;
    float *h_A, *h_B, *h_C;
    hipError_t err_;
};

template<int BLKX, int BLKY>
void MultiSizeTest(std::vector<float> &time) {
    time.clear();

    {
        TestGemm<BLKX, BLKY> test(256, 256, 256);
        test.InitHost();
        test.InitDevice();
        test.WarmUp();
        time.push_back(test.Run(10));
        test.Free();
    }

    {
        TestGemm<BLKX, BLKY> test(512, 512, 512);
        test.InitHost();
        test.InitDevice();
        test.WarmUp();
        time.push_back(test.Run(10));
        test.Free();
    }

    {
        TestGemm<BLKX, BLKY> test(1024, 1024, 1024);
        test.InitHost();
        test.InitDevice();
        test.WarmUp();
        time.push_back(test.Run(10));
        test.Free();
    }

    {
        TestGemm<BLKX, BLKY> test(2048, 2048, 2048);
        test.InitHost();
        test.InitDevice();
        test.WarmUp();
        time.push_back(test.Run(10));
        test.Free();
    }

    {
        TestGemm<BLKX, BLKY> test(4096, 4096, 4096);
        test.InitHost();
        test.InitDevice();
        test.WarmUp();
        time.push_back(test.Run(10));
        test.Free();
    }
    std::cout << std::endl << BLKY << "x" << BLKX << " " << "test results" << std::endl;
    std::cout << "256     512     1024     2048     4096" << std::endl;
    std::cout << "---------------------------------------" << std::endl;
    for (int i = 0; i < time.size(); ++i) {
        std::cout << std::setw(10) << std::setprecision(2) << time[i];
    }
    std::cout << std::endl;
}

int main() {

    std::vector<float> time;
    MultiSizeTest<16, 16>(time);
}

template<int BLKX, int BLKY>
void TestGemm<BLKX, BLKY>::InitHost() {
    h_A = new float[M * K];
    h_B = new float[K * N];
    h_C = new float[M * N];

    for (auto i = 0; i < M * K; ++i) {
        h_A[i] = 1.0f;
    }

    for (auto i = 0; i < K * N; ++i) {
        h_B[i] = 1.0f;
    }

    for (auto i = 0; i < M * N; ++i) {
        h_C[i] = 0.0f;
    }
}

template<int BLKX, int BLKY>
void TestGemm<BLKX, BLKY>::InitDevice() {
    if ((err_ = hipMalloc((void **) &d_A, M * K * sizeof(float))) != hipSuccess) {
        std::cout << "Failed to allocate device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }

    if ((err_ = hipMalloc((void **) &d_B, K * N * sizeof(float))) != hipSuccess) {
        std::cout << "Failed to allocate device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }

    if ((err_ = hipMalloc((void **) &d_C, M * N * sizeof(float))) != hipSuccess) {
        std::cout << "Failed to allocate device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }

    if ((err_ = hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cout << "Failed to copy dato to device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }

    if ((err_ = hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cout << "Failed to copy dato to device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }

    if ((err_ = hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cout << "Failed to copy dato to device memory: "
                  << hipGetErrorString(err_) << std::endl;
        exit(EXIT_FAILURE);
    }
}

template<int BLKX, int BLKY>
void TestGemm<BLKX, BLKY>::Free() {
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

template<int BLKX, int BLKY>
void TestGemm<BLKX, BLKY>::WarmUp() {

    std::cout << "matrix size: M = " << M << ", N = " << N << ", K = " << K << std::endl;
    std::cout << "grid size: Z = " << grid.z << ", Y = " << grid.y << ", X = " << grid.x << std::endl;
    std::cout << "block size: Z = " << block.z << ", Y = " << block.y << ", X = " << block.x << std::endl;
    std::cout << " ----------- warmup() ----------- " << std::endl;
    cu_sgemm<<<grid, block>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < M * N; ++i) {
        if (fabs(h_C[i] - 1.0f * K) > 1e-6) {
            std::cout << "error in sgemm" << std::endl;
        }
    }
    std::cout << "pass" << std::endl;
}
template<int BLKX, int BLKY>
float TestGemm<BLKX, BLKY>::Run(int n_iter) {
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    std::cout << " ----------- run test() ----------- " << std::endl;

    // 记录开始时刻的时间戳
    hipEventRecord(start, 0);
    // Do Something

    for (int i = 0; i < n_iter; ++i) {
        cu_sgemm<<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }

    // 记录结束时刻的时间戳
    hipEventRecord(stop, 0);
    // 等待事件同步值
    hipEventSynchronize(stop);

    // 根据开始和结束时刻的时间戳，计算其中所经过的时间
    hipEventElapsedTime(&elapsedTime, start, stop);
    // 打印时间
    printf("Average run time: %6.2f ms\n", elapsedTime / float(n_iter));

    return elapsedTime / float(n_iter);
}